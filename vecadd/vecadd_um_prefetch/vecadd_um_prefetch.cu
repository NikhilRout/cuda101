#include "hip/hip_runtime.h"
//Sum of two N-elements vectors using Unified Memory and Prefetching
//doing sin^2 + cos^2 = 1
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//check to compensate for FP rep inaccuracy
const double epsilon = 1e-6; 

__global__ void vecAdd(double *a, double *b, double *c, int N){
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(int argc, char* argv[]){

    int N = 1 << 16; //default value: 2^16 = 65536

    //Input vector size argument handling
    if(argc == 1){
        printf("Usage: %s <vector_size>\n", argv[0]);
        printf("Using default size: %d\n", N);
    } else if(argc == 2){
        N = atoi(argv[1]);
        if(N <= 0){
            printf("Error: Vector size must be positive\n");
            return 1;
        }
        printf("Using vector size: %d\n", N);
    } else{
        printf("Error: Too many arguments\n");
        printf("Usage: %s <vector_size>\n", argv[0]);
        return 1;
    }

    //common host and device vectors (unified memory pointers)
    double *a, *b, *c;

    //size of each vector (in bytes)
    size_t bytes = N * sizeof(double);

    //Alocating unified memory for pointers
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    //Getting device ID for prefetching calls
    int gpu_id;
    hipGetDevice(&gpu_id);

    //Setting hints about data and prefetching
    hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, gpu_id);

    //Initializing input vectors to sin^2 and cos^2
    for(int i = 0; i < N; ++i){
        a[i] = sin(i) * sin(i);
        b[i] = cos(i) * cos(i);
    }

    //Prefetching 'a' and 'b' arrays to device
    hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, gpu_id);
    hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, gpu_id);
    hipMemPrefetchAsync(a, bytes, gpu_id);
    hipMemPrefetchAsync(b, bytes, gpu_id);

    //number of threads per thread block (blockSize)
    int NUM_THREADS = 1 << 10; //1024

    //number of thread blocks in grid (gridSize)
    //pad extra thread block to grid if N isnt perfectly divisible by NUM_THREADS
    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    //alternatively int NUM_BLOCKS = ceil((float)n/NUM_THREADS)

    //Launching kernel on device
    //kernel calls are async --> host programs continues execution after call
    vecAdd<<<NUM_BLOCKS, NUM_THREADS>>>(a, b, c, N);

    //wait for all previous operations to complete before using values
    //need to explicitly do this cause we dont get the implicit sync of hipMemcpy
    hipDeviceSynchronize();

    //Prefetching to host
    hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

    //Result verification
    double sum = 0;
    for(int i = 0; i < N; ++i)
        sum += c[i];
    sum /= (double)N;
    if(fabs(sum - 1.0) < epsilon)
        printf("PASS\n");
    else
        printf("FAIL\n");
    
    //Free-ing unified memory
    hipFree(a);
    hipFree(b);
    hipFree(c);
}
