//doing sin^2 + cos^2 = 1

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//check to compensate for FP rep inaccuracy
const double epsilon = 1e-6; 

__global__ void vecAdd(double *a, double *b, double *c, int N){
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(int argc, char* argv[]){

    int N = 1 << 16; //default value: 2^16 = 65536

    //Input vector size argument handling
    if(argc == 1){
        printf("Usage: %s <vector_size>\n", argv[0]);
        printf("Using default size: %d\n", N);
    } else if(argc == 2){
        N = atoi(argv[1]);
        if(N <= 0){
            printf("Error: Vector size must be positive\n");
            return 1;
        }
        printf("Using vector size: %d\n", N);
    } else{
        printf("Error: Too many arguments\n");
        printf("Usage: %s <vector_size>\n", argv[0]);
        return 1;
    }

    //host vectors
    double *h_a, *h_b, *h_c;

    //device vectors
    double *d_a, *d_b, *d_c;

    //size of each vector (in bytes)
    size_t bytes = N * sizeof(double);

    //Allocating pinned memory on host for host vectors
    hipHostMalloc(&h_a, bytes, hipHostMallocDefault); //alternatively: h_a = (double*)malloc(bytes);
    hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_c, bytes, hipHostMallocDefault);
    
    //Allocating memory on device for device vectors
    hipMalloc(&d_a,bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    //Initializing input vectors to sin^2 and cos^2
    for(int i = 0; i < N; ++i){
        h_a[i] = sin(i) * sin(i);
        h_b[i] = cos(i) * cos(i);
    }

    //Copying input data from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    //number of threads per thread block (blockSize)
    int NUM_THREADS = 1 << 8; //256

    //number of thread blocks in grid (gridSize)
    //pad extra thread block to grid if N isnt perfectly divisible by NUM_THREADS
    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    //alternatively int NUM_BLOCKS = ceil((float)n/NUM_THREADS)

    //Launching kernel on device
    //kernel calls are async --> host programs continues execution after call
    vecAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

    //Copying output data from device to host
    //cudaMemcpy is a synchronous operation --> waits for kernel call to complete
    //acts as both memcpy + sync barrier
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    //Result verification
    double sum = 0;
    for(int i = 0; i < N; ++i)
        sum += h_c[i];
    sum /= (double)N;
    if(fabs(sum - 1.0) < epsilon)
        printf("PASS\n");
    else
        printf("FAIL\n");
    
    //Releasing device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    //Releasing host memory
    free(h_a);
    free(h_b);
    free(h_c);
}
